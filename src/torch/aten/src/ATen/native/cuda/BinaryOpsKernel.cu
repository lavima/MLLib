#include "hip/hip_runtime.h"
#include <ATen/Context.h>
#include <ATen/Dispatch.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/BinaryOps.h>
#include <limits>


// NOTE: CUDA on Windows requires that the enclosing function
// of a __device__ lambda not have internal linkage.

namespace at { namespace native {

void add_kernel_cuda(TensorIterator& iter, Scalar alpha_scalar) {
  AT_DISPATCH_ALL_TYPES_AND2(kHalf, kBool, iter.dtype(), "add_cuda/sub_cuda", [&]() {
    auto alpha = alpha_scalar.to<scalar_t>();
    gpu_kernel_with_scalars(iter, [alpha]GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
      return a + alpha * b;
    });
  });
}

static void sub_kernel_cuda(TensorIterator& iter, Scalar alpha_scalar) {
  add_kernel_cuda(iter, -alpha_scalar);
}

void div_kernel_cuda(TensorIterator& iter) {
  if (!isIntegralType(iter.dtype(), /*includeBool*/ false) && iter.is_cpu_scalar(2)) {
    // optimization for floating-point types: if the second operand is a CPU
    // scalar, compute a * reciprocal(b). Note that this may lose one bit of
    // precision compared to computing the division.
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "div_cuda", [&]() {
      auto inv_b = scalar_t(1.0 / iter.scalar_value<scalar_t>(2));
      iter.remove_operand(2);
      gpu_kernel(iter, [inv_b]GPU_LAMBDA(scalar_t a) -> scalar_t {
        return a * inv_b;
      });
    });
  } else {
    AT_DISPATCH_ALL_TYPES_AND(kHalf, iter.dtype(), "div_cuda", [&]() {
      gpu_kernel_with_scalars(iter, []GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
        return a / b;
      });
    });
  }
}

void mul_kernel_cuda(TensorIterator& iter) {
  if (iter.dtype() == ScalarType::Bool) {
    // Workaround for the error: '*' in boolean context, suggest '&&' instead [-Werror=int-in-bool-context]
    gpu_kernel_with_scalars(iter, []GPU_LAMBDA(bool a, bool b) -> bool {
      return a && b;
    });
  } else {
    AT_DISPATCH_ALL_TYPES_AND(kHalf, iter.dtype(), "mul_cuda", [&]() {
      gpu_kernel_with_scalars(iter, []GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
        return a * b;
      });
    });
  }
}

void atan2_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "atan2_cuda", [&]() {
    gpu_kernel_with_scalars(iter, []GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
      return ::atan2(a, b);
    });
  });
}

void logical_xor_kernel_cuda(TensorIterator& iter) {
  if (iter.dtype() == ScalarType::Bool) {
    AT_DISPATCH_ALL_TYPES_AND2(kHalf, kBool, iter.input_dtype(), "logical_xor_cuda", [&]() {
      gpu_kernel_with_scalars(iter, []GPU_LAMBDA(scalar_t a, scalar_t b) -> bool {
        return bool(a) != bool(b);
      });
    });
  } else {
    AT_DISPATCH_ALL_TYPES_AND(kHalf, iter.dtype(), "logical_xor_cuda", [&]() {
      gpu_kernel_with_scalars(iter, []GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
        return static_cast<scalar_t>(bool(a) != bool(b));
      });
    });
  }
}

void lt_kernel_cuda(TensorIterator& iter) {
  if (iter.dtype() == ScalarType::Bool) {
    AT_DISPATCH_ALL_TYPES_AND2(kHalf, kBool, iter.input_dtype(), "lt_cpu", [&]() {
      gpu_kernel_with_scalars(iter, []GPU_LAMBDA(scalar_t a, scalar_t b) -> bool {
        return a < b;
      });
    });
  } else {
    AT_DISPATCH_ALL_TYPES_AND(kHalf, iter.dtype(), "lt_cpu", [&]() {
      gpu_kernel_with_scalars(iter, []GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
        return a < b;
      });
    });
  }
}

void le_kernel_cuda(TensorIterator& iter) {
  if (iter.dtype() == ScalarType::Bool) {
    AT_DISPATCH_ALL_TYPES_AND2(kHalf, kBool, iter.input_dtype(), "le_cpu", [&]() {
      gpu_kernel_with_scalars(iter, []GPU_LAMBDA(scalar_t a, scalar_t b) -> bool {
        return a <= b;
      });
    });
  } else {
    AT_DISPATCH_ALL_TYPES_AND(kHalf, iter.dtype(), "le_cpu", [&]() {
      gpu_kernel_with_scalars(iter, []GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
        return a <= b;
      });
    });
  }
}

void gt_kernel_cuda(TensorIterator& iter) {
  if (iter.dtype() == ScalarType::Bool) {
    AT_DISPATCH_ALL_TYPES_AND2(kHalf, kBool, iter.input_dtype(), "gt_cpu", [&]() {
      gpu_kernel_with_scalars(iter, []GPU_LAMBDA(scalar_t a, scalar_t b) -> bool {
        return a > b;
      });
    });
  } else {
    AT_DISPATCH_ALL_TYPES_AND(kHalf, iter.dtype(), "gt_cpu", [&]() {
      gpu_kernel_with_scalars(iter, []GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
        return a > b;
      });
    });
  }
}

void ge_kernel_cuda(TensorIterator& iter) {
  if (iter.dtype() == ScalarType::Bool) {
    AT_DISPATCH_ALL_TYPES_AND2(kHalf, kBool, iter.input_dtype(), "ge_cpu", [&]() {
      gpu_kernel_with_scalars(iter, []GPU_LAMBDA(scalar_t a, scalar_t b) -> bool {
        return a >= b;
      });
    });
  } else {
    AT_DISPATCH_ALL_TYPES_AND(kHalf, iter.dtype(), "ge_cpu", [&]() {
      gpu_kernel_with_scalars(iter, []GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
        return a >= b;
      });
    });
  }
}

void eq_kernel_cuda(TensorIterator& iter) {
  if (iter.dtype() == ScalarType::Bool) {
    AT_DISPATCH_ALL_TYPES_AND2(kHalf, kBool, iter.input_dtype(), "eq_cpu", [&]() {
      gpu_kernel_with_scalars(iter, []GPU_LAMBDA(scalar_t a, scalar_t b) -> bool {
        return a == b;
      });
    });
  } else {
    AT_DISPATCH_ALL_TYPES_AND(kHalf, iter.dtype(), "eq_cpu", [&]() {
      gpu_kernel_with_scalars(iter, []GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
        return a == b;
      });
    });
  }
}

void ne_kernel_cuda(TensorIterator& iter) {
  if (iter.dtype() == ScalarType::Bool) {
    AT_DISPATCH_ALL_TYPES_AND2(kHalf, kBool, iter.input_dtype(), "ne_cpu", [&]() {
      gpu_kernel_with_scalars(iter, []GPU_LAMBDA(scalar_t a, scalar_t b) -> bool {
        return a != b;
      });
    });
  } else {
    AT_DISPATCH_ALL_TYPES_AND(kHalf, iter.dtype(), "ne_cpu", [&]() {
      gpu_kernel_with_scalars(iter, []GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
        return a != b;
      });
    });
  }
}

void smooth_l1_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_ALL_TYPES_AND(kHalf, iter.dtype(), "smooth_l1_cuda", [&]() {
    gpu_kernel(iter, [] GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
      auto z = fabs(a - b);
      return z < scalar_t(1.) ? scalar_t(0.5) * z * z : z - scalar_t(0.5);
    });
  });
}

REGISTER_DISPATCH(add_stub, &add_kernel_cuda);
REGISTER_DISPATCH(sub_stub, &sub_kernel_cuda);
REGISTER_DISPATCH(div_stub, &div_kernel_cuda);
REGISTER_DISPATCH(mul_stub, &mul_kernel_cuda);
REGISTER_DISPATCH(atan2_stub, &atan2_kernel_cuda);
REGISTER_DISPATCH(logical_xor_stub, &logical_xor_kernel_cuda);
REGISTER_DISPATCH(lt_stub, &lt_kernel_cuda);
REGISTER_DISPATCH(le_stub, &le_kernel_cuda);
REGISTER_DISPATCH(gt_stub, &gt_kernel_cuda);
REGISTER_DISPATCH(ge_stub, &ge_kernel_cuda);
REGISTER_DISPATCH(eq_stub, &eq_kernel_cuda);
REGISTER_DISPATCH(ne_stub, &ne_kernel_cuda);
REGISTER_DISPATCH(smooth_l1_stub, &smooth_l1_kernel_cuda);

}} // namespace at::native
