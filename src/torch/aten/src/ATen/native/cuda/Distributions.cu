#include "hip/hip_runtime.h"
#include <ATen/Dispatch.h>
#include <ATen/ExpandUtils.h>
#include <ATen/NativeFunctions.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/AccumulateType.h>
#include <ATen/CUDAGenerator.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand_kernel.h>
#include <utility>
#include <functional>

#include <ATen/native/Distributions.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/TensorIterator.h>
#include <ATen/LegacyTHFunctionsCUDA.h>

#include <THC/THCGeneral.h>
#include <THC/THCApply.cuh>
#include <THC/THCDeviceUtils.cuh>

#include <cstdint>
#include <limits>
#include <utility>
#include <type_traits>

/**
 * Note [Register spilling in hiprand call for CUDA < 10]
 * ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
 * For CUDA < 10, hiprandStatePhilox4_32_10_t engine achieves poor performance (60% SOL bandwidth)
 * when called to generate one random number at a time. This is because the line
 *            unsigned ret = (&state->output.x)[state->STATE++];
 * in
 *            QUALIFIERS unsigned int hiprand(hiprandStatePhilox4_32_10_t *state)
 * in hiprand/hiprand_kernel.h dynamically indexes into state.output, preventing the compiler from ever
 * storing state.output in registers.
 *
 * CUDA 10 fixed this problem. However, for backwards compatibility, in the following kernels
 * we are using hiprand distributions that utilize curand4 call. curand4 call doesn't have the
 * register spilling problem.
 */

namespace {

// launch bounds used for kernels utilizing TensorIterator
const uint32_t block_size_bound = 256;
const uint32_t grid_size_bound = 4;
// number of randoms given by distributions like hiprand_uniform4, hiprand_uniform2_double
// used in calculating philox offset.
const uint32_t curand4_engine_calls = 4;

// utility function that calculates proper philox_offset
// for distributions utilizing TensorIterator. For distributions using
// TensorIterator, we are using a grid-stride loop with each
// thread yielding one element per thread. For the edge of the grid-stride
// loop, if the tensor size is large, the unroll loop will kick in and the float4
// from curand4 will start getting utilized (for common tensor sizes, we end up
// using rand.x from each thread). Hence, the philox_offset is
// (number of elements per thread * number of engine calls), which makes
// sure that philox offset increment is not less than the number of randoms used
// in each thread.
std::tuple<uint64_t, dim3, dim3> calc_execution_policy(int64_t total_elements) {
  const uint64_t numel = static_cast<uint64_t>(total_elements);
  const uint32_t block_size = block_size_bound;
  const uint32_t unroll = curand4_engine_calls;
  dim3 dim_block(block_size);
  dim3 grid((numel + block_size - 1) / block_size);
  uint32_t blocks_per_sm = at::cuda::getCurrentDeviceProperties()->maxThreadsPerMultiProcessor / block_size;
  grid.x = std::min(
      static_cast<uint32_t>(at::cuda::getCurrentDeviceProperties()->multiProcessorCount) * blocks_per_sm,
      grid.x);
  //number of times random will be generated per thread, to offset philox counter in thc random state
  uint64_t counter_offset = ((numel - 1) / (block_size * grid.x * unroll) + 1)
                                * curand4_engine_calls;
  return std::make_tuple(counter_offset, grid, dim_block);
}

// grid stride loop kernel for distributions
template<typename accscalar_t, int unroll_factor, typename dist_t, typename transform_t>
C10_LAUNCH_BOUNDS_2(block_size_bound, grid_size_bound)
__global__ void distribution_elementwise_grid_stride_kernel(int numel,
                                                            std::pair<uint64_t, uint64_t> seeds,
                                                            const dist_t dist_func,
                                                            const transform_t transform_func) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  hiprandStatePhilox4_32_10_t state;
  hiprand_init(
      seeds.first,
      idx,
      seeds.second,
      &state);
  int rounded_size = ((numel - 1)/(blockDim.x * gridDim.x * unroll_factor)+1) *
      blockDim.x * gridDim.x * unroll_factor;
  for(int linear_index = idx; linear_index < rounded_size; linear_index += blockDim.x * gridDim.x * unroll_factor) {
    auto rand = dist_func(&state);
    #pragma unroll
    for (int ii = 0; ii < unroll_factor; ii++) {
      int li = linear_index + blockDim.x * gridDim.x * ii;
      if (li < numel) {
        transform_func(li, static_cast<accscalar_t>((&rand.x)[ii]));
      }
    }
    __syncthreads();
  }
}

/**
 * distribution_nullary_kernel is analogous to gpu_kernel in
 * ATen/native/cuda/Loops.cuh. Like gpu_kernel, it uses
 * TensorIterator to launch a kernel. However, the differences are
 *   - it launches a grid-stride loop based kernel. The kernel is not
 *     generic like elementwise_kernel in Loops.cuh and is specialized
 *     for the distribution kernels here.
 *   - For big size tensors, we can launch multiple kernels recursively
 *     (i.e. if (!iter.can_use_32bit_indexing())) and hence, the philox
 *     offset calculation is done in this function.
 *
 * FIXME: Can we specialize elementwise_kernel and launch_kernel in Loops.cuh
 * to have grid-stride loop kernel and then use that to launch our distribution
 * kernels? Note that we need a grid-stride loop kernel because, we found by testing
 * that it achieves peak effective bandwidth.
 */
template<typename scalar_t,
         typename accscalar_t,
         int unroll_factor,
         typename dist_t,
         typename transform_t>
void distribution_nullary_kernel(at::TensorIterator& iter,
                                 at::CUDAGenerator* gen,
                                 const dist_t& dist_func,
                                 const transform_t transform_func) {
  static_assert(unroll_factor >= 1, "unroll_factor must be >= 1.");
  int64_t numel = iter.numel();
  if (numel == 0) {
    return;
  }

  auto execution_policy = calc_execution_policy(numel);
  auto counter_offset = std::get<0>(execution_policy);
  auto grid = std::get<1>(execution_policy);
  auto block = std::get<2>(execution_policy);
  std::pair<uint64_t, uint64_t> rng_engine_inputs;
  {
    // See Note [Acquire lock when using random generators]
    std::lock_guard<std::mutex> lock(gen->mutex_);
    rng_engine_inputs = gen->philox_engine_inputs(counter_offset);
  }

  if (!iter.can_use_32bit_indexing()) {
    for (auto& sub_iter : iter.with_32bit_indexing()) {
      distribution_nullary_kernel<scalar_t, accscalar_t, unroll_factor>(sub_iter,
        gen, dist_func, transform_func);
    }
    return;
  }

  char* out_data = (char*)iter.data_ptr(0);

  auto stream = at::cuda::getCurrentCUDAStream();
  if (iter.is_trivial_1d()) {
    auto strides = iter.get_inner_strides();
    int stride0 = strides[0];
    distribution_elementwise_grid_stride_kernel<accscalar_t, unroll_factor><<<grid, block, 0, stream>>>(
      numel,
      rng_engine_inputs,
      dist_func,
      [=]__device__(int idx, accscalar_t rand) {
        scalar_t* out = (scalar_t*)&out_data[stride0 * idx];
        *out = transform_func(rand);
      }
    );
  } else {
    auto offset_calc = at::native::make_offset_calculator<1>(iter);
    distribution_elementwise_grid_stride_kernel<accscalar_t, unroll_factor><<<grid, block, 0, stream>>>(
      numel,
      rng_engine_inputs,
      dist_func,
      [=]__device__(int idx, accscalar_t rand) {
        auto offsets = offset_calc.get(idx);
        scalar_t* out = (scalar_t*)&out_data[offsets[0]];
        *out = transform_func(rand);
      }
    );
  }
  AT_CUDA_CHECK(hipGetLastError());
}

template <typename scalar_t>
void poisson_cuda_kernel(
    at::Tensor& ret,
    const at::Tensor& lambda,
    std::pair<uint64_t, uint64_t> seeds) {
  at::cuda::CUDA_tensor_apply2<scalar_t, scalar_t>(
      ret,
      lambda,
      [seeds] __device__(
          scalar_t & ret_val, const scalar_t& lambda) {
        hiprandStatePhilox4_32_10_t state;
        hiprand_init(
            seeds.first,
            blockIdx.x * blockDim.x + threadIdx.x,
            seeds.second,
            &state);
        ret_val = static_cast<scalar_t>(hiprand_poisson(&state, lambda));
      });
}

template <typename scalar_t>
void gamma_cuda_kernel(
    at::Tensor& ret,
    const at::Tensor& alpha,
    std::pair<uint64_t, uint64_t> seeds) {
  using accscalar_t = at::acc_type<scalar_t, true>;
  at::cuda::CUDA_tensor_apply2<scalar_t, scalar_t>(
      ret,
      alpha,
      [seeds] __device__(
          scalar_t & ret_val, const scalar_t& alpha) {
        hiprandStatePhilox4_32_10_t state;
        hiprand_init(
            seeds.first,
            blockIdx.x * blockDim.x + threadIdx.x,
            seeds.second,
            &state);

        auto uniform_lambda = [&state] __device__ () {
          return hiprand_uniform(&state);
        };
        BaseSampler<accscalar_t, decltype(uniform_lambda)> standard_uniform(uniform_lambda);

        auto normal_lambda = [&state] __device__ () {
          return hiprand_normal(&state);
        };
        BaseSampler<accscalar_t, decltype(normal_lambda)> standard_normal(normal_lambda);
        auto sample = sample_gamma<scalar_t, accscalar_t, decltype(uniform_lambda), decltype(normal_lambda)>(alpha, standard_uniform, standard_normal);
        auto min_value = std::numeric_limits<scalar_t>::min();
        ret_val = (min_value > sample) ? min_value : sample;
      });
}

template <typename scalar_t>
void gamma_grad_cuda_kernel(
    at::Tensor& ret,
    const at::Tensor& self,
    const at::Tensor& output) {
  using accscalar_t = at::acc_type<scalar_t, true>;
  at::cuda::CUDA_tensor_apply3<scalar_t, scalar_t, scalar_t>(
      ret, self, output,
      [] __device__ (scalar_t& ret_val, const scalar_t& self_val, const scalar_t &output_val) {
        ret_val = standard_gamma_grad_one<scalar_t, accscalar_t>(self_val, output_val);
      });
}

template <typename scalar_t>
void dirichlet_grad_cuda_kernel(
    at::Tensor& ret,
    const at::Tensor& x,
    const at::Tensor& alpha,
    const at::Tensor& total) {
  using accscalar_t = at::acc_type<scalar_t, true>;
  at::cuda::CUDA_tensor_apply4<scalar_t, scalar_t, scalar_t, scalar_t>(
      ret, x, alpha, total,
      [] __device__ (scalar_t& ret_val, const scalar_t& x_val, const scalar_t& alpha_val, const scalar_t& total_val) {
        ret_val = dirichlet_grad_one<scalar_t, accscalar_t>(x_val, alpha_val, total_val);
      });
}

template<typename scalar_t, typename prob_t>
void bernoulli_tensor_cuda_kernel(
    at::Tensor& ret, const at::Tensor& p,
    std::pair<uint64_t, uint64_t> seeds) {
  // The template argument `4` below indicates that we want to operate on four
  // element at each time. See NOTE [ CUDA_tensor_applyN helpers ] for details.
  at::cuda::CUDA_tensor_apply2<scalar_t, prob_t, 4>(
      ret, p,
      [seeds] __device__(
          int n, scalar_t& v1, scalar_t& v2, scalar_t& v3, scalar_t& v4,
          const prob_t& p1, const prob_t& p2, const prob_t& p3, const prob_t& p4) {
        hiprandStatePhilox4_32_10_t state;
        hiprand_init(
            seeds.first,
            blockIdx.x * blockDim.x + threadIdx.x,
            seeds.second,
            &state);
        // See Note [Register spilling in hiprand call for CUDA < 10]
        float4 rand = hiprand_uniform4(&state);
        switch (n) {
          case 4: {
            assert(0 <= p4 && p4 <= 1);
            v4 = static_cast<scalar_t>(rand.w <= p4);
            // fallthrough
          }
          case 3: {
            assert(0 <= p3 && p3 <= 1);
            v3 = static_cast<scalar_t>(rand.z <= p3);
            // fallthrough
          }
          case 2: {
            assert(0 <= p2 && p2 <= 1);
            v2 = static_cast<scalar_t>(rand.y <= p2);
            // fallthrough
          }
          case 1: {
            assert(0 <= p1 && p1 <= 1);
            v1 = static_cast<scalar_t>(rand.x <= p1);
          }
        }
      }
    );
}

template<typename scalar_t>
void dirichlet_scalar_cuda_kernel(
    at::Tensor& ret,
    const at::Tensor& gamma) {
  auto gamma_sum = gamma.sum(-1, true).expand(ret.sizes());
  at::cuda::CUDA_tensor_apply3<scalar_t, scalar_t, scalar_t>(ret, gamma, gamma_sum,
  [] __device__(scalar_t &ret_val, const scalar_t &gamma, const scalar_t &gamma_sum) {
    ret_val = gamma / gamma_sum;
    auto min_value = std::numeric_limits<scalar_t>::min();
    auto max_value = 1 - std::numeric_limits<scalar_t>::epsilon();
    ret_val = (min_value > ret_val) ? min_value : ret_val;
    ret_val = (max_value < ret_val) ? max_value : ret_val;
  });
}

} // namespace

namespace at { namespace native {

Tensor _s_poisson_cuda(const Tensor& lambda, Generator* gen_) {
  auto gen = get_generator_or_default<CUDAGenerator>(gen_, cuda::detail::getDefaultCUDAGenerator());
  std::pair<uint64_t, uint64_t> rng_engine_inputs;
  {
    // See Note [Acquire lock when using random generators]
    std::lock_guard<std::mutex> lock(gen->mutex_);
    rng_engine_inputs = gen->philox_engine_inputs(20);
  }
  Tensor ret = at::empty(lambda.sizes(), lambda.options());
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(ret.scalar_type(), "poisson_cuda", [&] {
    poisson_cuda_kernel<scalar_t>(ret, lambda, rng_engine_inputs);
  });
  return ret;
}

Tensor _s_gamma_cuda(const Tensor& alpha, Generator* gen_) {
  auto gen = get_generator_or_default<CUDAGenerator>(gen_, cuda::detail::getDefaultCUDAGenerator());
  std::pair<uint64_t, uint64_t> rng_engine_inputs;
  {
    // See Note [Acquire lock when using random generators]
    std::lock_guard<std::mutex> lock(gen->mutex_);
    rng_engine_inputs = gen->philox_engine_inputs(10);
  }
  Tensor ret = at::empty(alpha.sizes(), alpha.options());
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(ret.scalar_type(), "gamma_cuda", [&] {
     gamma_cuda_kernel<scalar_t>(ret, alpha, rng_engine_inputs);
   });
  return ret;
}

Tensor _s_dirichlet_cuda(const Tensor& alpha, Generator* gen_) {
  auto gen = get_generator_or_default<CUDAGenerator>(gen_, cuda::detail::getDefaultCUDAGenerator());
  std::pair<uint64_t, uint64_t> rng_engine_inputs;
  {
    // See Note [Acquire lock when using random generators]
    std::lock_guard<std::mutex> lock(gen->mutex_);
    rng_engine_inputs = gen->philox_engine_inputs(10);
  }
  Tensor ret = at::empty(alpha.sizes(), alpha.options());
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(ret.scalar_type(), "dirichlet", [&] {
    Tensor gamma = at::empty(alpha.sizes(), alpha.options());
    gamma_cuda_kernel<scalar_t>(gamma, alpha, rng_engine_inputs);
    dirichlet_scalar_cuda_kernel<scalar_t>(ret, gamma);
  });
  return ret;
}

Tensor _standard_gamma_grad_cuda(const Tensor& self, const Tensor& output) {
  Tensor ret = at::empty(self.sizes(), self.options());
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(self.scalar_type(), "_standard_gamma_grad_cuda", [&] {
     gamma_grad_cuda_kernel<scalar_t>(ret, self, output);
   });
  return ret;
}

Tensor _dirichlet_grad_cuda(const Tensor& x, const Tensor& alpha, const Tensor& total) {
  Tensor ret = at::empty(x.sizes(), x.options());
  AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "_dirichlet_grad_cuda", [&] {
    dirichlet_grad_cuda_kernel<scalar_t>(ret, x, alpha, total);
  });
  return ret;
}

Tensor& bernoulli_tensor_cuda_(Tensor &self, const Tensor& p_, Generator* gen_) {
#ifdef BUILD_NAMEDTENSOR
  NoNamesGuard guard;
#endif
  auto gen = get_generator_or_default<CUDAGenerator>(gen_, cuda::detail::getDefaultCUDAGenerator());
  std::pair<uint64_t, uint64_t> rng_engine_inputs;
  {
    // See Note [Acquire lock when using random generators]
    std::lock_guard<std::mutex> lock(gen->mutex_);
    rng_engine_inputs = gen->philox_engine_inputs(10);
  }
  auto p = std::get<0>(expand_inplace(self, p_.to(kCUDA)));
  AT_DISPATCH_ALL_TYPES_AND2(
    at::ScalarType::Half, at::ScalarType::Bool, self.scalar_type(), "bernoulli_tensor_cuda_self_", [&] {
      using self_t = scalar_t;
      AT_DISPATCH_FLOATING_TYPES_AND_HALF(p.scalar_type(), "bernoulli_tensor_cuda_p_", [&] {
        using p_t = scalar_t;
        return bernoulli_tensor_cuda_kernel<self_t, p_t>(self, p, rng_engine_inputs);
      });
   });
  return self;
}

void uniform_kernel_cuda(TensorIterator& iter, double from_, double to_, Generator* gen_) {
  auto gen = get_generator_or_default<CUDAGenerator>(gen_, cuda::detail::getDefaultCUDAGenerator());
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "uniform_cuda", [&] {
    auto from = static_cast<scalar_t>(from_);
    auto to = static_cast<scalar_t>(to_);
    TORCH_CHECK(from <= to,
      "uniform_ expects to return a [from, to) range, but found from=", from,
      " > to=", to);
    TORCH_CHECK((to - from) <= std::numeric_limits<scalar_t>::max(),
          "uniform_ expects to-from <= std::numeric_limits<", toString(iter.dtype()),
          ">::max(), but found to=", to, " and from=", from,
          " which result in to-from to exceed the limit");

    using accscalar_t = at::acc_type<scalar_t, true>;
    auto range = static_cast<accscalar_t>(to-from);
    from = static_cast<accscalar_t>(from);
    // define lambda to reverse bounds, multiply 'range' and add 'from_'
    auto uniform_func = [range, from] __device__ (accscalar_t rand) {
      // reverse the bounds of curand4 from (0, 1] to [0, 1)
      // Note that this method is from legacy THCTensorRandom and is likely to give
      // you more 0-s, since, the probability of gettings 1-s is higher than 0-s and
      // by reversing the bounds, we are flipping the probabilities of 1-s and 0-s.
      auto reverse_bound_rand = rand == static_cast<accscalar_t>(1.0) ? static_cast<accscalar_t>(0.0) : rand;
      return static_cast<scalar_t>(reverse_bound_rand * range + from);
    };
    if (std::is_same<scalar_t, double>::value) {
      distribution_nullary_kernel<scalar_t, accscalar_t, curand4_engine_calls/2>(iter,
        gen,
        [] __device__ (hiprandStatePhilox4_32_10_t* state) { return hiprand_uniform2_double(state); },
        uniform_func);
    } else {
      distribution_nullary_kernel<scalar_t, accscalar_t, curand4_engine_calls>(iter,
        gen,
        [] __device__ (hiprandStatePhilox4_32_10_t* state) { return hiprand_uniform4(state); },
        uniform_func);
    }
   });
}

void random_kernel_cuda(TensorIterator& iter, uint64_t range, int64_t base, Generator* gen_) {
  auto gen = get_generator_or_default<CUDAGenerator>(gen_, cuda::detail::getDefaultCUDAGenerator());
  AT_DISPATCH_ALL_TYPES_AND2(at::ScalarType::Bool, at::ScalarType::Half, iter.dtype(), "random_cuda", [&] {
    if (std::is_same<scalar_t, double>::value || std::is_same<scalar_t, int64_t>::value) {
      // define lambda to mod with range and add base
      auto random_func = [range, base] __device__ (uint64_t rand) {
        return static_cast<int64_t>(rand % range + base);
      };
      distribution_nullary_kernel<scalar_t, uint64_t, curand4_engine_calls/2>(iter,
        gen,
        [] __device__ (hiprandStatePhilox4_32_10_t* state) -> ulonglong2 {
          ulonglong2 ret;
          uint4 rand_val = curand4(state);
          ret.x = (static_cast<uint64_t>(rand_val.x) << 32) | rand_val.y;
          ret.y = (static_cast<uint64_t>(rand_val.z) << 32) | rand_val.w;
          return ret;
        },
        random_func);
    } else {
      auto random_func = [range, base] __device__ (uint32_t rand) {
        return static_cast<int32_t>(rand % static_cast<uint32_t>(range) + static_cast<int32_t>(base));
      };
      distribution_nullary_kernel<scalar_t, uint32_t, curand4_engine_calls>(iter,
        gen,
        [] __device__ (hiprandStatePhilox4_32_10_t* state) {
          return curand4(state);
        },
        random_func);
    }
   });
}

void normal_kernel_cuda(TensorIterator& iter, double mean_, double std_, Generator* gen_) {
  auto gen = get_generator_or_default<CUDAGenerator>(gen_, cuda::detail::getDefaultCUDAGenerator());
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "normal_cuda", [&] {
    using accscalar_t = at::acc_type<scalar_t, true>;
    auto mean = static_cast<accscalar_t>(mean_);
    auto std = static_cast<accscalar_t>(std_);
    // define lambda to multiply std and add mean
    auto normal_func = [mean, std] __device__ (accscalar_t rand) {
      return static_cast<scalar_t>(rand * std + mean);
    };
    if (std::is_same<scalar_t, double>::value) {
      distribution_nullary_kernel<scalar_t, accscalar_t, curand4_engine_calls/2>(iter,
        gen,
        [] __device__ (hiprandStatePhilox4_32_10_t* state) { return hiprand_normal2_double(state); },
        normal_func);
    } else {
      distribution_nullary_kernel<scalar_t, accscalar_t, curand4_engine_calls>(iter,
        gen,
        [] __device__ (hiprandStatePhilox4_32_10_t* state) { return hiprand_normal4(state); },
        normal_func);
    }
   });
}

void cauchy_kernel_cuda(TensorIterator& iter, double median_, double sigma_, Generator* gen_) {
  auto gen = get_generator_or_default<CUDAGenerator>(gen_, cuda::detail::getDefaultCUDAGenerator());
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "cauchy_cuda", [&] {
    using accscalar_t = at::acc_type<scalar_t, true>;
    auto median = static_cast<accscalar_t>(median_);
    auto sigma = static_cast<accscalar_t>(sigma_);
    if (std::is_same<scalar_t, double>::value) {
      // define lambda for cauchy transformation
      auto cauchy_func = [median, sigma] __device__ (accscalar_t rand) {
        return static_cast<scalar_t>(median + sigma *
                ::tan(static_cast<accscalar_t>(M_PI) * (rand-static_cast<accscalar_t>(0.5))));
      };
      distribution_nullary_kernel<scalar_t, accscalar_t, curand4_engine_calls/2>(iter,
        gen,
        [] __device__ (hiprandStatePhilox4_32_10_t* state) { return hiprand_uniform2_double(state); },
        cauchy_func);
    } else {
      // use __tanf fast approximation for peak bandwidth
      auto cauchy_func = [median, sigma] __device__ (accscalar_t rand) {
        return static_cast<scalar_t>(median + sigma *
                __tanf(static_cast<accscalar_t>(M_PI) * (rand-static_cast<accscalar_t>(0.5))));
      };
      distribution_nullary_kernel<scalar_t, accscalar_t, curand4_engine_calls>(iter,
        gen,
        [] __device__ (hiprandStatePhilox4_32_10_t* state) { return hiprand_uniform4(state); },
        cauchy_func);
    }
   });
}

void exponential_kernel_cuda(TensorIterator& iter, double lambda_, Generator* gen_) {
  auto gen = get_generator_or_default<CUDAGenerator>(gen_, cuda::detail::getDefaultCUDAGenerator());
  // Note that HIP doesn't support std::nextafter in device code.
  auto nextafter_1_0_float = std::nextafter(1.0f, 0.0f);
  auto nextafter_1_0_double = std::nextafter(1.0, 0.0);
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "exponential_cuda", [&] {
    using accscalar_t = at::acc_type<scalar_t, true>;
    auto lambda = static_cast<accscalar_t>(lambda_);
    if (std::is_same<scalar_t, double>::value) {
      // define lambda for exponential transformation
      auto exponential_func = [lambda, nextafter_1_0_double] __device__ (accscalar_t rand) {
        accscalar_t sample;
        // hiprand_uniform has (0,1] bounds. log(1) is 0 and exponential excludes 0.
        // Hence, squash the 1 to just below 1.
        if(rand == static_cast<accscalar_t>(1.0)) {
          sample = ::log(nextafter_1_0_double);
        } else {
          sample = ::log(rand);
        }
        return static_cast<scalar_t>(static_cast<accscalar_t>(-1.0) / lambda * sample);
      };
      distribution_nullary_kernel<scalar_t, accscalar_t, curand4_engine_calls/2>(iter,
        gen,
        [] __device__ (hiprandStatePhilox4_32_10_t* state) { return hiprand_uniform2_double(state); },
        exponential_func);
    } else {
      // use __logf fast approximation for peak bandwidth
      auto exponential_func = [lambda, nextafter_1_0_float] __device__ (accscalar_t rand) {
        accscalar_t sample;
        if(rand == static_cast<accscalar_t>(1.0)) {
          sample = __logf(nextafter_1_0_float);
        } else {
          sample = __logf(rand);
        }
        return static_cast<scalar_t>(static_cast<accscalar_t>(-1.0) / lambda * sample);
      };
      distribution_nullary_kernel<scalar_t, accscalar_t, curand4_engine_calls>(iter,
        gen,
        [] __device__ (hiprandStatePhilox4_32_10_t* state) { return hiprand_uniform4(state); },
        exponential_func);
    }
   });
}

void geometric_kernel_cuda(TensorIterator& iter, double p_, Generator* gen_) {
  auto gen = get_generator_or_default<CUDAGenerator>(gen_, cuda::detail::getDefaultCUDAGenerator());
  AT_DISPATCH_ALL_TYPES_AND(at::ScalarType::Half, iter.dtype(), "geometric_cuda", [&] {
    if (std::is_same<scalar_t, double>::value) {
      // define lambda for geometric transformation
      auto geometric_func = [p_] __device__ (double rand) {
        return static_cast<scalar_t>(::ceil(::log(rand) / ::log(static_cast<double>(1.0)-p_)));
      };
      distribution_nullary_kernel<scalar_t, double, curand4_engine_calls/2>(iter,
        gen,
        [] __device__ (hiprandStatePhilox4_32_10_t* state) { return hiprand_uniform2_double(state); },
        geometric_func);
    } else {
      auto p = static_cast<float>(p_);
      auto geometric_func = [p] __device__ (float rand) {
        // use __logf fast approximation for peak bandwidth
        return static_cast<scalar_t>(::ceil(__logf(rand) / __logf(static_cast<float>(1.0)-p)));
      };
      distribution_nullary_kernel<scalar_t, float, curand4_engine_calls>(iter,
        gen,
        [] __device__ (hiprandStatePhilox4_32_10_t* state) { return hiprand_uniform4(state); },
        geometric_func);
    }
   });
}

void log_normal_kernel_cuda(TensorIterator& iter, double mean_, double std_, Generator* gen_) {
  auto gen = get_generator_or_default<CUDAGenerator>(gen_, cuda::detail::getDefaultCUDAGenerator());
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "log_normal_cuda", [&] {
    using accscalar_t = at::acc_type<scalar_t, true>;
    auto mean = static_cast<accscalar_t>(mean_);
    auto std = static_cast<accscalar_t>(std_);
    if (std::is_same<scalar_t, double>::value) {
      // define lambda for log_normal transformation
      auto log_normal_func = [mean, std] __device__ (accscalar_t rand) {
        return static_cast<scalar_t>(::exp(rand * std + mean));
      };
      distribution_nullary_kernel<scalar_t, accscalar_t, curand4_engine_calls/2>(iter,
        gen,
        [] __device__ (hiprandStatePhilox4_32_10_t* state) { return hiprand_normal2_double(state); },
        log_normal_func);
    } else {
      auto log_normal_func = [mean, std] __device__ (accscalar_t rand) {
        // use __expf fast approximation for peak bandwidth
        return static_cast<scalar_t>(__expf(rand * std + mean));
      };
      distribution_nullary_kernel<scalar_t, accscalar_t, curand4_engine_calls>(iter,
        gen,
        [] __device__ (hiprandStatePhilox4_32_10_t* state) { return hiprand_normal4(state); },
        log_normal_func);
    }
   });
}

void bernoulli_scalar_cuda_kernel(TensorIterator& iter, double p_, Generator* gen_) {
  auto gen = get_generator_or_default<CUDAGenerator>(gen_, cuda::detail::getDefaultCUDAGenerator());
  AT_DISPATCH_ALL_TYPES_AND2(
    at::ScalarType::Half, at::ScalarType::Bool, iter.dtype(), "bernoulli_scalar_cuda_", [&] {
      if (std::is_same<scalar_t, double>::value) {
      // define lambda for bernoulli transformation
      auto bernoulli_func = [p_] __device__ (double rand) {
        return static_cast<scalar_t>(rand <= p_);
      };
      distribution_nullary_kernel<scalar_t, double, curand4_engine_calls/2>(iter,
        gen,
        [] __device__ (hiprandStatePhilox4_32_10_t* state) { return hiprand_uniform2_double(state); },
        bernoulli_func);
    } else {
      auto p = static_cast<float>(p_);
      auto bernoulli_func = [p] __device__ (float rand) {
        return static_cast<scalar_t>(rand <= p);
      };
      distribution_nullary_kernel<scalar_t, float, curand4_engine_calls>(iter,
        gen,
        [] __device__ (hiprandStatePhilox4_32_10_t* state) { return hiprand_uniform4(state); },
        bernoulli_func);
    }
   });
}

Tensor& uniform_cuda_(Tensor& self, double from, double to, Generator* gen) {
  auto iter = TensorIterator::nullary_op(self);
  uniform_kernel_cuda(iter, from, to, gen);
  return self;
}

Tensor& random_cuda_(Tensor& self, Generator* gen) {
  auto iter = TensorIterator::nullary_op(self);
  uint64_t range;
  auto iter_scalar_type = iter.dtype();
  if (isFloatingType(iter_scalar_type)) {
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter_scalar_type, "random_cuda_range_calc", [&] {
      range = static_cast<uint64_t>((1ULL << std::numeric_limits<scalar_t>::digits) + 1);
    });
  } else {
    AT_DISPATCH_INTEGRAL_TYPES(iter_scalar_type, "random_cuda_range_calc", [&] {
      range = static_cast<uint64_t>(std::numeric_limits<scalar_t>::max()) + 1;
    });
  }
  random_kernel_cuda(iter, range, 0, gen);
  return self;
}

Tensor& clamped_random_cuda_(Tensor& self, int64_t from, int64_t to, Generator* gen) {
  TORCH_CHECK(from < to, "random_ expects 'from' to be less than 'to', but got from=", from, " >= to=", to);
  auto iter = TensorIterator::nullary_op(self);
  uint64_t range = to - from;
  random_kernel_cuda(iter, range, from, gen);
  return self;
}

Tensor& capped_random_cuda_(Tensor& self, int64_t to, Generator* gen) {
  return clamped_random_cuda_(self, 0, to, gen);
}

Tensor& normal_cuda_(Tensor& self, double mean, double std, Generator* gen) {
  TORCH_CHECK(std > 0.0, "normal_ expects std > 0.0, but found std=", std);
  auto iter = TensorIterator::nullary_op(self);
  normal_kernel_cuda(iter, mean, std, gen);
  return self;
}

Tensor& normal_out_cuda(Tensor& output, const Tensor& mean, double std, Generator* gen) {
  normal_cuda_(output, 0, std, gen);
  output.add_(mean);
  return output;
}

Tensor& normal_out_cuda(Tensor& output, double mean, const Tensor& std, Generator* gen) {
  normal_cuda_(output, 0, 1, gen);
  auto mean_tensor = at::full({1}, mean, output.options());
  // NB: addcmul_out copies the tensor to be added into the output.
  // Please look at aten/src/THC/generic/THCTensorMathPointwise.cu
  // The previous function here was addcmul_out(output, mean_tensor, output, std, 1);
  // The third argument is not a constant reference and hence the samples in output are overwritten.
  // Consequently, the computation performed is mean_tensor + mean_tensor * std instead of mean_tensor + output * std
  output.mul_(std).add_(mean_tensor);
  return output;
}

Tensor& normal_out_cuda(Tensor& output, const Tensor& mean, const Tensor& std, Generator* gen) {
  normal_cuda_(output, 0, 1, gen);
  // NB: addcmul_out copies the tensor to be added into the output.
  // Please look at aten/src/THC/generic/THCTensorMathPointwise.cu
  // The previous function here was addcmul_out(output, mean, output, std, 1);
  // The third argument is not a constant reference and hence the samples in output are overwritten.
  // Consequently, the computation performed is mean + mean * std instead of mean + output * std
  output.mul_(std).add_(mean);
  return output;
}

Tensor normal_cuda(const Tensor& mean, double std, Generator* gen) {
  Tensor ret = at::empty_like(mean);
  normal_out_cuda(ret, mean, std, gen);
  return ret;
}

Tensor normal_cuda(double mean, const Tensor& std, Generator* gen) {
  Tensor ret = at::empty_like(std);
  normal_out_cuda(ret, mean, std, gen);
  return ret;
}

Tensor normal_cuda(const Tensor& mean, const Tensor& std, Generator* gen) {
  Tensor ret = at::empty_like(mean);
  normal_out_cuda(ret, mean, std, gen);
  return ret;
}

Tensor& cauchy_cuda_(Tensor& self, double median, double sigma, Generator* gen) {
  auto iter = TensorIterator::nullary_op(self);
  cauchy_kernel_cuda(iter, median, sigma, gen);
  return self;
}

Tensor& exponential_cuda_(Tensor& self, double lambda, Generator* gen) {
  auto iter = TensorIterator::nullary_op(self);
  exponential_kernel_cuda(iter, lambda, gen);
  return self;
}

Tensor& geometric_cuda_(Tensor& self, double p, Generator* gen) {
  TORCH_CHECK(0 < p && p < 1, "geometric_ expects p to be in (0, 1), but got p=", p);
  auto iter = TensorIterator::nullary_op(self);
  geometric_kernel_cuda(iter, p, gen);
  return self;
}

Tensor& log_normal_cuda_(Tensor& self, double mean, double std, Generator* gen) {
  TORCH_CHECK(std > 0.0, "log_normal_ expects std > 0.0, but found std=", std);
  auto iter = TensorIterator::nullary_op(self);
  log_normal_kernel_cuda(iter, mean, std, gen);
  return self;
}

Tensor& bernoulli_scalar_cuda_(Tensor &self, double p, Generator* gen) {
  TORCH_CHECK(0 <= p && p <= 1, "bernoulli_ expects p to be in [0, 1], but got p=", p);
  auto iter = TensorIterator::nullary_op(self);
  bernoulli_scalar_cuda_kernel(iter, p, gen);
  return self;
}

}} // namespace at::native
