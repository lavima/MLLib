#include "hip/hip_runtime.h"
#include <ATen/native/Indexing.h>
#include <ATen/native/IndexingUtils.h>

#include <ATen/ATen.h>
#include <ATen/NativeFunctions.h>
#include <ATen/ExpandUtils.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/AccumulateType.h>

#include <THC/THCDeviceUtils.cuh>
#include <THC/THCGeneral.h>
#include <THC/THCTensorSort.cuh>
#include <ATen/cuda/HIPContext.h>
#include <THC/THCThrustAllocator.cuh>
#include <thrust/execution_policy.h>
#include <thrust/sort.h>

#include <c10/macros/Macros.h>

namespace {

template <typename scalar_t, int SZ>
__global__ void indexing_backward_kernel(
  int64_t* sorted_indices, int64_t* indices, scalar_t* grad_output, scalar_t* grad_weight,
  int64_t numel, int64_t stride, int64_t stride_before, int64_t outer_dim) {
//numel is total number of flattened indices, not expanded to dimensions that are not indexed.
//stride is the cumulative size of the not-indexed last dimensions
//stride_before is the stride of the dimension immediately preceding first indexed dimension
//if indexing starts from the 0th dimension, stride_before does not matter because blockIdx.z will be 0 in this case
//outer_dim is number of elements in the first unindexed dimensions
  using accscalar_t = at::acc_type<scalar_t, true>;

  // Each warp is responsible for an input into the LookupTable.
  // If the preceding input has the same destination index as this input, then the warp
  // exits immediately. The warp also processes subsequent inputs with the
  // same value.
  //
  // Input Warp
  // 1     <warp 1>
  // 1     <warp 1> (<warp 2> exits without doing any work)
  // 5     <warp 3>
  // 8     <warp 4>

  // Number of values processed by each thread (grain size)
  for (int z = blockIdx.z; z < outer_dim; z += gridDim.z){
    int idx = blockIdx.x * blockDim.y + threadIdx.y;
    if (idx < numel
        && (idx == 0 || sorted_indices[idx] != sorted_indices[idx - 1])){
      do {
        int start_feature = threadIdx.x + blockIdx.y * blockDim.x * SZ;
        const int weight_row = ((int) sorted_indices[idx]) * stride + z * stride_before;
        const int grad_row = ((int) indices[idx]) * stride + z * numel * stride;
        const accscalar_t scale = (accscalar_t)1.0;

        accscalar_t gradient[SZ];
        accscalar_t weight[SZ];

        while (start_feature < stride) {
          #pragma unroll
          for (int ii = 0; ii < SZ; ii++) {
            int feature_dim = start_feature + ii * C10_WARP_SIZE;
            if (feature_dim < stride) {
              gradient[ii] = static_cast<accscalar_t>(grad_output[grad_row + feature_dim]);
              weight[ii] = static_cast<accscalar_t>(grad_weight[weight_row + feature_dim]);
            }
          }

          #pragma unroll
          for (int ii = 0; ii < SZ; ii++) {
            weight[ii] += gradient[ii] * scale;
          }

          #pragma unroll
          for (int ii = 0; ii < SZ; ii++) {
            int feature_dim = start_feature + ii * C10_WARP_SIZE;
            if (feature_dim < stride) {
                grad_weight[weight_row + feature_dim] = static_cast<scalar_t>(weight[ii]);
            }
          }
          start_feature += gridDim.y * blockDim.x * SZ;
        }

        idx++;
      } while (idx < numel && sorted_indices[idx] == sorted_indices[idx - 1]);
    }
  }
}


}    


namespace at { namespace native {

static Tensor wrapIndexOnce(const Tensor & index, int64_t dim, int64_t dim_size, bool check_range=true) {
//we don't need to check range in backward - if there were out of bounds indices forward should already have errored out 
  if (index.numel() != 0 && check_range) {
    auto max_idx = index.max().item<int64_t>();
    auto min_idx = index.min().item<int64_t>();
    if (max_idx >= dim_size) {
      AT_INDEX_ERROR("index ", max_idx, " is out of bounds for dimension ", dim, " with size ", dim_size);
    }
    if (min_idx < -dim_size) {
      AT_INDEX_ERROR("index ", min_idx, " is out of bounds for dimension ", dim, " with size ", dim_size);
    }
  }
  return index.remainder(dim_size);
}

static std::vector<int64_t> computeLinearStride(const Tensor & tensor) {
  // computes the stride as if tensor were contigous
  auto sizes = tensor.sizes();
  std::vector<int64_t> stride(tensor.dim());
  stride[tensor.dim() - 1] = 1;
  std::partial_sum(sizes.rbegin(), sizes.rend() - 1, stride.rbegin() + 1, std::multiplies<int64_t>());
  return stride;
}

static std::tuple<Tensor, int64_t, int64_t, int64_t> 
computeLinearIndex(const Tensor & src, TensorList indices, bool check_range) {
  auto strides = computeLinearStride(src);
  const auto& backend = src.type().backend();

  // Compute the linear index by multiplying the indexing tensors by the
  // stride and summing them. All the indexing tensors have the same shape at
  // this point. We also compute the number of dimensions before and after that
  // are not being index.
  Tensor linearIndex;
  int64_t emptyBefore = 0, emptyAfter = 0, nElemBefore = 1, nElemAfter = 1, strideBefore =0;
  for (auto i = decltype(src.dim()){0}; i < src.dim(); i++) {
    if (indices[i].defined()) {
      // Cast index to the longType matching src's backend
      // This allows us to support ie indexing a cuda tensor with a cpu tensor
      Tensor index = (wrapIndexOnce(indices[i], i, src.size(i), check_range) * strides[i]).toBackend(backend);
      if (linearIndex.defined()) {
        linearIndex += index;
      } else {
        linearIndex = index;
        if (i>0) {
           strideBefore = src.stride(i-1); // stride after undefined dimensions
        }
      }
    } else if (linearIndex.defined()) {
      emptyAfter++;
      nElemAfter *= src.size(i);
    } else {
      emptyBefore++;
      nElemBefore *= src.size(i);
    }
  }

  return std::make_tuple(std::move(linearIndex), nElemBefore, strideBefore, nElemAfter);
}


static std::tuple<Tensor, Tensor, int64_t, int64_t, int64_t, std::vector<int64_t>> makeLinearIndex(Tensor self, TensorList orig, bool check_range) {
  checkIndexTensorTypes(orig);
  // first expand BoolTensor (masks) or ByteTensor (masks) into 1 or more LongTensors
  auto indices = expandTensors(self, orig);
  // next broadcast all index tensors together
  indices = expand_outplace(indices);
  // add missing null Tensors so that it matches self.dim()
  while (indices.size() < (size_t)self.dim()) {
    indices.emplace_back();
  }
  // if the non-null indices are not all adjacent, transpose self and indices
  // together so that they're adjacent at the front
  std::vector<int64_t> inversePerm;
  if (!hasContiguousSubspace(indices)) {
    std::tie(self, indices, inversePerm) = transposeToFrontAndInvPerm(self, indices);
  }
  int64_t nElemBefore, strideBefore, nElemAfter;
  Tensor linearIndex;
  std::tie(linearIndex, nElemBefore, strideBefore, nElemAfter) = computeLinearIndex(self, indices, check_range);
  return std::make_tuple(linearIndex, self, nElemBefore, strideBefore, nElemAfter, inversePerm);
}


namespace {
void index_put_accum_kernel(Tensor & self, TensorList indices, const Tensor & value, bool unsafe) {
  if (indices.size() > (size_t)self.dim()) {
    AT_INDEX_ERROR("too many indices for tensor of dimension ", self.dim(), " (got ", indices.size(), ")");
  }
  auto value_ = value.contiguous();
  Tensor linearIndex, expandedValue, src;
  int64_t nElemBefore, strideBefore, sliceSize;
  std::vector<int64_t> inversePerm;
  std::tie(linearIndex, src, nElemBefore, strideBefore, sliceSize, inversePerm) = makeLinearIndex(self, indices, !unsafe);
  int64_t num_indices = linearIndex.numel();
  if (num_indices > 0 && sliceSize > 0) {
      const bool permuted = !src.is_contiguous();
      auto src_ = permuted ? src.contiguous() : src;
      linearIndex = linearIndex.view(-1);
      auto sorted_indices = at::empty_like(linearIndex);
      auto orig_indices = at::empty_like(linearIndex);
      using device_ptr = thrust::device_ptr<int64_t>;
      const hipStream_t stream = at::cuda::getCurrentCUDAStream();
    
      linearIndex.div_(sliceSize);
      {
      sorted_indices.copy_(linearIndex);
      auto allocator = THCThrustAllocator(globalContext().lazyInitCUDA());
      auto policy = thrust::cuda::par(allocator).on(stream);
    
      // Fill sortedOrigIndices with sequential indices
      const auto count_iter = thrust::counting_iterator<int64_t>(0);
      auto orig_data = device_ptr(orig_indices.data_ptr<int64_t>());
      thrust::copy(policy, count_iter, count_iter + num_indices, orig_data);
    
      // Sort the inputs into sorted with the corresponding indices; we
      // don't need a stable or multidimensional sort, so just use Thrust
      // directly
      // Sort; a stable sort is not required
      // NB - not passing comparator causes thrust to use radix sort, and it hurts perf A LOT, at least for medium (few K) sized indices
      auto sorted_data = device_ptr(sorted_indices.data_ptr<int64_t>());
      thrust::sort_by_key(policy, sorted_data, sorted_data + num_indices, orig_data, ThrustLTOp<int64_t>());
      }
      TORCH_INTERNAL_ASSERT(linearIndex.numel()*sliceSize*nElemBefore == value.numel(), "number of flattened indices did not match number of elements in the value tensor", linearIndex.numel()*sliceSize*nElemBefore, value.numel());
      TORCH_CHECK(self.numel() < std::numeric_limits<int>::max(), "index_put_ with accumulation is not supported on large tensors, number of source elements =", self.numel(), "file a support request on github");
      TORCH_CHECK(value.numel() < std::numeric_limits<int>::max(), "index_put_ with accumulation is not supported on large tensors, number of source elements =", value.numel(), "file a support request on github");
      const int UNROLL = 4;
      const int indices_per_block = 4;
      dim3 grid(THCCeilDiv(num_indices, (int64_t) indices_per_block),
           std::min<int>(at::cuda::getCurrentDeviceProperties()->maxGridSize[1], THCCeilDiv(sliceSize, (int64_t) (C10_WARP_SIZE*UNROLL))),
           std::min(std::max<int>(1,nElemBefore), at::cuda::getCurrentDeviceProperties()->maxGridSize[2]));
      dim3 block(C10_WARP_SIZE, indices_per_block);
  
      AT_DISPATCH_FLOATING_TYPES_AND_HALF(value_.scalar_type(), "embedding_backward", [&] {
      indexing_backward_kernel<scalar_t, UNROLL><<<grid, block, 0, stream>>>(
        sorted_indices.data_ptr<int64_t>(),
        orig_indices.data_ptr<int64_t>(),
        value_.data_ptr<scalar_t>(),
        src_.data_ptr<scalar_t>(),
        num_indices,
        sliceSize,
        strideBefore,
        nElemBefore);
      });
      THCudaCheck(hipGetLastError());
      if (permuted)
          self.copy_(src_.permute(inversePerm));
  }
}

REGISTER_CUDA_DISPATCH(index_put_accum_stub, &index_put_accum_kernel);
} //anonymous
} //at
} //native


