#include "hip/hip_runtime.h"
#include <ATen/Context.h>
#include <ATen/Dispatch.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/Pow.h>

namespace at { namespace native {

namespace {

template <typename T>
static inline __host__ __device__ T powi(T a, T b) {
  T result = 1;
  while (b) {
    if (b & 1) {
       result *= a;
    }
    b /= 2;
    a *= a;
  }
  return result;
}

template <typename T>
static inline __host__ __device__ T sqrt(T x) {
  return std::sqrt(x);
}

void pow_tensor_tensor_kernel(TensorIterator& iter) {
  if (isFloatingType(iter.dtype())) {
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "pow_cuda", [&]() {
      gpu_kernel(iter, []GPU_LAMBDA(scalar_t base, scalar_t exp) -> scalar_t {
        return std::pow(base, exp);
      });
    });
  } else {
    AT_DISPATCH_INTEGRAL_TYPES(iter.dtype(), "pow_cuda", [&]() {
      gpu_kernel(iter, []GPU_LAMBDA(scalar_t base, scalar_t exp) -> scalar_t {
        return powi(base, exp);
      });
    });
  }
}

template<typename Base_type, typename Exp_type>
void pow_tensor_scalar_kernel_impl(TensorIterator& iter,
                                                 Exp_type exp) {
  const auto d_exp = static_cast<double>(exp);
  if (d_exp == 0.5) {
    gpu_kernel(iter, [=]GPU_LAMBDA(Base_type base) -> Base_type {
      return ::sqrt(base);
    });
  } else if (d_exp == 2) {
    gpu_kernel(iter, [=]GPU_LAMBDA(Base_type base) -> Base_type {
      return base * base;
    });
  } else if (d_exp == 3) {
    gpu_kernel(iter, [=]GPU_LAMBDA(Base_type base) -> Base_type {
      return base * base * base;
    });
  } else if (d_exp == -0.5) {
    gpu_kernel(iter, [=]GPU_LAMBDA(Base_type base) -> Base_type {
      return 1.0 / ::sqrt(base);
    });
  } else if (d_exp == -1) {
    gpu_kernel(iter, [=]GPU_LAMBDA(Base_type base) -> Base_type {
      return 1.0 / base;
    });
  } else if (d_exp == -2) {
    gpu_kernel(iter, [=]GPU_LAMBDA(Base_type base) -> Base_type {
      return 1.0 / (base * base);
    });
  } else {
    gpu_kernel(iter, [=]GPU_LAMBDA(Base_type base) -> Base_type {
      return std::pow(base, exp);
    });
  }
}

void pow_tensor_scalar_kernel(TensorIterator& iter, Scalar exp_scalar) {
  if (isFloatingType(iter.dtype()) || exp_scalar.isIntegral(false)) {
    AT_DISPATCH_ALL_TYPES_AND(kHalf, iter.dtype(), "pow_cuda", [&]() {
      const auto exp = exp_scalar.to<scalar_t>();
      pow_tensor_scalar_kernel_impl<scalar_t>(iter, exp);
    });
  } else {
    const auto exp = exp_scalar.to<float>();
    AT_DISPATCH_INTEGRAL_TYPES(iter.dtype(), "pow_cuda", [&]() {
      pow_tensor_scalar_kernel_impl<scalar_t>(iter, exp);
    });
  }
}

} // anonymous namespace

REGISTER_DISPATCH(pow_tensor_tensor_stub, &pow_tensor_tensor_kernel);
REGISTER_DISPATCH(pow_tensor_scalar_stub, &pow_tensor_scalar_kernel);

}} // namespace at::native
