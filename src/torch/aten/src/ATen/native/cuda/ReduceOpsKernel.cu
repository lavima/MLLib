#include "hip/hip_runtime.h"
#include <ATen/native/SharedReduceOps.h>
#include <ATen/AccumulateType.h>
#include <ATen/Context.h>
#include <ATen/Dispatch.h>
#include <ATen/cuda/NumericLimits.cuh>
#include <ATen/native/cuda/DeviceSqrt.cuh>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/cuda/Reduce.cuh>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/ReduceOps.h>
#include <limits>
#include <tuple>
#include <THC/THCNumerics.cuh>
#include <thrust/tuple.h>
#include <thrust/pair.h>


namespace at { namespace native {

template <typename scalar_t, typename acc_t=scalar_t, typename out_t=scalar_t>
void sum_kernel_impl(TensorIterator& iter) {
  gpu_reduce_kernel<scalar_t, out_t>(iter, func_wrapper<out_t> ([]GPU_LAMBDA(acc_t a, acc_t b) -> acc_t {
    return a + b;
  }));
}

template <typename scalar_t>
void std_var_kernel_impl(TensorIterator& iter, bool unbiased, bool take_sqrt) {
  // reducing unrolling factor to 2 for welford kernel
  // This is necessary to lower register usage that leads to register spills.
  gpu_reduce_kernel<scalar_t, scalar_t, 2>(iter, WelfordOps<scalar_t, scalar_t, int32_t, float, thrust::tuple<scalar_t, scalar_t>> { unbiased, take_sqrt }, WelfordData<scalar_t, int32_t, float> {});
}

template <>
void std_var_kernel_impl<at::Half>(TensorIterator& iter, bool unbiased, bool take_sqrt) {
  // reducing unrolling factor to 2 for welford kernel
  // This is necessary to lower register usage that leads to register spills.
  gpu_reduce_kernel<at::Half, at::Half, 2>(iter, WelfordOps<at::Half, float, int32_t, float, thrust::tuple<at::Half, at::Half>> { unbiased, take_sqrt }, WelfordData<float, int32_t, float> {});
}

template <typename scalar_t, typename acc_t=scalar_t>
void prod_kernel_impl(TensorIterator& iter) {
  gpu_reduce_kernel<scalar_t, scalar_t>(iter, func_wrapper<scalar_t> ([]GPU_LAMBDA(acc_t a, acc_t b) -> acc_t {
    return a * b;
  }), 1);
}

static void std_var_kernel_cuda(TensorIterator& iter, bool unbiased, bool take_sqrt) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "std", [&]() {
    std_var_kernel_impl<scalar_t>(iter, unbiased, take_sqrt);
  });
}

template <typename scalar_t, typename acc_t=scalar_t, typename out_t=scalar_t>
void mean_kernel_impl(TensorIterator& iter) {
  float factor = float(iter.num_output_elements()) / iter.numel();
  gpu_reduce_kernel<scalar_t, out_t>(iter, MeanOps<acc_t, float> {factor});
}

template <typename scalar_t, typename acc_t=scalar_t, typename out_t=scalar_t>
void norm_kernel_cuda_impl(TensorIterator& iter, Scalar val) {
  float p;
  if (val.isIntegral(false)) {
     p = val.to<int64_t>();
  } else if (val.isFloatingPoint()) {
     p = val.to<acc_t>();
  } else {
     AT_ERROR("norm_kernel_cuda_impl expects norm to be integer or float");
  }

  if (p == static_cast<float>(0)) {
    gpu_reduce_kernel<scalar_t, out_t>(iter, NormZeroOps<acc_t>(), 0);
  } else if (p == static_cast<float>(1)) {
    gpu_reduce_kernel<scalar_t, out_t>(iter, NormOneOps<acc_t>(), 0);
  } else if (p == static_cast<float>(INFINITY)) {
    gpu_reduce_kernel<scalar_t, out_t>(iter, AbsMaxOps<acc_t>(), std::numeric_limits<acc_t>::min());
  } else if (p == static_cast<float>(-INFINITY)) {
    gpu_reduce_kernel<scalar_t, out_t>(iter, AbsMinOps<acc_t>(), std::numeric_limits<acc_t>::max());
  } else {
    gpu_reduce_kernel<scalar_t, out_t>(iter, NormOps<acc_t>{ acc_t(p) }, 0);
  }
}

static void sum_kernel_cuda(TensorIterator& iter) {
  if (iter.dtype() == kHalf) {
    return sum_kernel_impl<at::Half, float>(iter);
  } else if (iter.dtype(1) == kHalf && iter.dtype() == kFloat) {
    // type promotion that does cast and reduction in a single kernel
    return sum_kernel_impl<at::Half, float, float>(iter);
  }
  AT_DISPATCH_ALL_TYPES_AND(ScalarType::Bool, iter.dtype(), "sum_cuda", [&]() {
    sum_kernel_impl<scalar_t>(iter);
  });
}

static void prod_kernel_cuda(TensorIterator& iter) {
  if (iter.dtype() == kHalf) {
    return prod_kernel_impl<at::Half, float>(iter);
  }
  AT_DISPATCH_ALL_TYPES(iter.dtype(), "prod_cuda", [&]() {
    prod_kernel_impl<scalar_t>(iter);
  });
}

static void mean_kernel_cuda(TensorIterator& iter) {
  if (iter.dtype() == kHalf) {
    return mean_kernel_impl<at::Half, float>(iter);
  } else if (iter.dtype(1) == kHalf && iter.dtype() == kFloat) {
    // type promotion that does cast and reduction in a single kernel
    return mean_kernel_impl<at::Half, float, float>(iter);
  }
  AT_DISPATCH_ALL_TYPES(iter.dtype(), "mean_cuda", [&]() {
    mean_kernel_impl<scalar_t>(iter);
  });
}

static void norm_kernel_cuda(TensorIterator& iter, Scalar p) {
  if (iter.dtype() == kHalf) {
    return norm_kernel_cuda_impl<at::Half, float>(iter, p);
  } else if (iter.dtype(1) == kHalf && iter.dtype() == kFloat) {
    // type promotion that does cast and reduction in a single kernel
    return norm_kernel_cuda_impl<at::Half, float, float>(iter, p);
  }
  AT_DISPATCH_FLOATING_TYPES(iter.dtype(), "norm_cuda", [&]() {
    norm_kernel_cuda_impl<scalar_t>(iter, p);
  });
}

void and_kernel_cuda(TensorIterator& iter) {
  gpu_reduce_kernel<uint8_t, uint8_t>(
    iter, func_wrapper<uint8_t> ([]GPU_LAMBDA(uint8_t a, uint8_t b) -> uint8_t {
      return a && b;
    }), true);
}

void or_kernel_cuda(TensorIterator& iter) {
  gpu_reduce_kernel<uint8_t, uint8_t>(
    iter, func_wrapper<uint8_t> ([]GPU_LAMBDA(uint8_t a, uint8_t b) -> uint8_t {
      return a || b;
    }), false);
}

template <typename scalar_t, typename acc_t=scalar_t>
void max_values_kernel_cuda_impl(TensorIterator& iter) {
  gpu_reduce_kernel<scalar_t, scalar_t>(
    iter, func_wrapper<acc_t> ([]GPU_LAMBDA(acc_t a, acc_t b) -> acc_t {
      return (THCNumerics<acc_t>::isnan(a) || a > b) ? a : b;
    }), at::numeric_limits<acc_t>::lower_bound());
}

template <typename scalar_t, typename acc_t=scalar_t>
void min_values_kernel_cuda_impl(TensorIterator& iter) {
  gpu_reduce_kernel<scalar_t, scalar_t>(
    iter, func_wrapper<acc_t> ([]GPU_LAMBDA(acc_t a, acc_t b) -> acc_t {
      return (THCNumerics<acc_t>::isnan(a) || a < b) ? a : b;
    }), at::numeric_limits<acc_t>::upper_bound());
}

void max_values_kernel_cuda(TensorIterator& iter) {
  if (iter.dtype(1) == kHalf) {
    max_values_kernel_cuda_impl<at::Half, float>(iter);
  } else {
    AT_DISPATCH_ALL_TYPES(iter.dtype(), "max_values_cuda", [&]() {
      max_values_kernel_cuda_impl<scalar_t>(iter);
    });
  }
}

void min_values_kernel_cuda(TensorIterator& iter) {
  if (iter.dtype(1) == kHalf) {
    min_values_kernel_cuda_impl<at::Half, float>(iter);
  } else {
    AT_DISPATCH_ALL_TYPES(iter.dtype(), "min_values_cuda", [&]() {
      min_values_kernel_cuda_impl<scalar_t>(iter);
    });
  }
}

template <typename scalar_t, typename acc_t=scalar_t>
void argmax_kernel_cuda_impl(TensorIterator& iter) {
  gpu_reduce_kernel<scalar_t, int64_t>(
    iter,
    ArgMaxOps<acc_t>{},
    thrust::pair<acc_t, int64_t>(at::numeric_limits<acc_t>::lower_bound(), 0));
};

template <typename scalar_t, typename acc_t=scalar_t>
void argmin_kernel_cuda_impl(TensorIterator& iter) {
  gpu_reduce_kernel<scalar_t, int64_t>(
    iter,
    ArgMinOps<acc_t>{},
    thrust::pair<acc_t, int64_t>(at::numeric_limits<acc_t>::upper_bound(), 0));
};

void argmax_kernel_cuda(TensorIterator& iter) {
  if (iter.dtype(1) == kHalf) {
    // Instead of implementing is_nan and warp_shfl_down
    // we can convert halves to float and do all the operations in float
    argmax_kernel_cuda_impl<at::Half, float>(iter);
  } else {
    AT_DISPATCH_ALL_TYPES(iter.dtype(1), "argmax_cuda", [&]() {
      argmax_kernel_cuda_impl<scalar_t>(iter);
    });
  }
}

void argmin_kernel_cuda(TensorIterator& iter) {
  if (iter.dtype(1) == kHalf) {
    // Instead of implementing is_nan and warp_shfl_down
    // we can convert halves to float and do all the operations in float
    argmin_kernel_cuda_impl<at::Half, float>(iter);
  } else {
    AT_DISPATCH_ALL_TYPES(iter.dtype(1), "argmin_cuda", [&]() {
      argmin_kernel_cuda_impl<scalar_t>(iter);
    });
  }
}

REGISTER_DISPATCH(std_var_stub, &std_var_kernel_cuda);
REGISTER_DISPATCH(sum_stub, &sum_kernel_cuda);
REGISTER_DISPATCH(prod_stub, &prod_kernel_cuda);
REGISTER_DISPATCH(mean_stub, &mean_kernel_cuda);
REGISTER_DISPATCH(norm_stub, &norm_kernel_cuda);
REGISTER_DISPATCH(and_stub, &and_kernel_cuda);
REGISTER_DISPATCH(or_stub, &or_kernel_cuda);
REGISTER_DISPATCH(max_values_stub, &max_values_kernel_cuda);
REGISTER_DISPATCH(min_values_stub, &min_values_kernel_cuda);
REGISTER_DISPATCH(argmax_stub, &argmax_kernel_cuda);
REGISTER_DISPATCH(argmin_stub, &argmin_kernel_cuda);

}} // namespace at::native
