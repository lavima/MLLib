#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>

#include <ATen/native/cuda/Resize.cuh>
#include <ATen/native/ResizeCommon.h>

namespace at { namespace native {

Tensor& resize_cuda_(Tensor& self, IntArrayRef size) {
#ifdef BUILD_NAMEDTENSOR
  if (self.has_names()) {
    return resize_named_tensor_(self, size);
  }
#endif
  auto* self_ = self.unsafeGetTensorImpl();
  resize_impl_cuda_(self_, size, /*strides=*/c10::nullopt);
  self_->maybe_zero_dim(size.size() == 0);
  return self;
}

}}
