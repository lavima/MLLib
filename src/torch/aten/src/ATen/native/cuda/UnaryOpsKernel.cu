#include "hip/hip_runtime.h"
#include <limits>
#include <ATen/native/UnaryOps.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/Context.h>
#include <ATen/Dispatch.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/cuda/Math.cuh>

namespace at { namespace native {

void bitwise_not_kernel_cuda(TensorIterator& iter) {
  if (iter.dtype() == ScalarType::Bool) {
    gpu_kernel(iter, []GPU_LAMBDA(bool a) {
      return !a;
    });
  } else {
    AT_DISPATCH_INTEGRAL_TYPES(iter.dtype(), "bitwise_not_cuda", [&]() {
      gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
        return ~a;
      });
    });
  }
}

void logical_not_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_ALL_TYPES_AND2(kBool, kHalf, iter.dtype(1), "logical_not_cuda", [&]() {
    using self_t = scalar_t;
    AT_DISPATCH_ALL_TYPES_AND2(kBool, kHalf, iter.dtype(0), "logical_not_cuda", [&]() {
      gpu_kernel(iter, []GPU_LAMBDA(self_t a) -> scalar_t { return static_cast<scalar_t>(!a); });
    });
  });
}

void asin_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "asin_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return ::asin(a);
    });
  });
}

void ceil_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "ceil_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return std::ceil(a);
    });
  });
}

void expm1_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "expm1_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return ::expm1(a);
    });
  });
}


void floor_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "floor_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return std::floor(a);
    });
  });
}

void log_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "log_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return std::log(a);
    });
  });
}

void log10_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "log10_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return ::log10(a);
    });
  });
}

void log1p_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "log1p_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return ::log1p(a);
    });
  });
}

void log2_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "log2_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return ::log2(a);
    });
  });
}

void neg_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_ALL_TYPES_AND(ScalarType::Half, iter.dtype(), "neg_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return -a;
    });
  });
}

// We manually overload nearbyint because std::nearbyint does not work with ROCm.
template <typename scalar_t>
__host__ __device__ static inline scalar_t nearbyint_wrapper(scalar_t a) {
  return static_cast<scalar_t>(::nearbyintf(static_cast<float>(a)));
}

__host__ __device__ static inline double nearbyint_wrapper(double a) {
  return ::nearbyint(a);
}

void round_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "round_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      // We do not use std::round because we would like to round midway numbers to the nearest even integer.
      return nearbyint_wrapper(a);
    });
  });
}

// We manually overload trunc because std::trunc does not work with ROCm.
template <typename scalar_t>
__host__ __device__ static inline scalar_t trunc_wrapper(scalar_t a) {
  return static_cast<scalar_t>(::truncf(static_cast<float>(a)));
}

__host__ __device__ static inline double trunc_wrapper(double a) {
  return ::trunc(a);
}

void trunc_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "trunc_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return trunc_wrapper(a);
    });
  });
}

void rsqrt_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "rsqrt_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      // In CUDA, ::rsqrt is overloaded for float and at::Half here is implicitly cast to float.
      return ::rsqrt(a);
    });
  });
}

void sign_kernel_cuda(TensorIterator& iter){
    if (iter.dtype() == ScalarType::Bool) {
      gpu_kernel(iter, []GPU_LAMBDA(bool a){
        return a;
      });
    } else {
      AT_DISPATCH_ALL_TYPES_AND(ScalarType::Half, iter.dtype(), "sign_cuda", [&]() {
          gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
              scalar_t zero = scalar_t(0);
              return (zero < a) - (a < zero);
          });
      });
    }
}

void sin_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "sin_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return ::sin(a);
    });
  });
}

void sinh_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "sinh_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return ::sinh(a);
    });
  });
}

void sqrt_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "sqrt_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return ::sqrt(a);
    });
  });
}

void erfinv_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "erfinv_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return ::erfinv(a);
    });
  });
}

void digamma_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "digamma_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return calc_digamma(a);
    });
  });
}

void trigamma_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "trigamma_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return calc_trigamma(a);
    });
  });
}

void polygamma_kernel_cuda(TensorIterator& iter, int64_t n) {
  switch (n) {
    case 0: digamma_kernel_cuda(iter); break;
    case 1: trigamma_kernel_cuda(iter); break;
    default: TORCH_CHECK(false, "polygamma(n,x) is not implemented for n>=2, but was ", n);
  }
}

void lgamma_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "lgamma_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return ::lgamma(a);
    });
  });
}

REGISTER_DISPATCH(bitwise_not_stub, &bitwise_not_kernel_cuda);
REGISTER_DISPATCH(logical_not_stub, &logical_not_kernel_cuda);
REGISTER_DISPATCH(asin_stub, &asin_kernel_cuda);
REGISTER_DISPATCH(ceil_stub, &ceil_kernel_cuda);
REGISTER_DISPATCH(expm1_stub, &expm1_kernel_cuda);
REGISTER_DISPATCH(floor_stub, &floor_kernel_cuda);
REGISTER_DISPATCH(log_stub, &log_kernel_cuda);
REGISTER_DISPATCH(log10_stub, &log10_kernel_cuda);
REGISTER_DISPATCH(log2_stub, &log2_kernel_cuda);
REGISTER_DISPATCH(log1p_stub, &log1p_kernel_cuda);
REGISTER_DISPATCH(neg_stub, &neg_kernel_cuda);
REGISTER_DISPATCH(round_stub, &round_kernel_cuda);
REGISTER_DISPATCH(rsqrt_stub, &rsqrt_kernel_cuda);
REGISTER_DISPATCH(sign_stub, &sign_kernel_cuda);
REGISTER_DISPATCH(sin_stub, &sin_kernel_cuda);
REGISTER_DISPATCH(sinh_stub, &sinh_kernel_cuda);
REGISTER_DISPATCH(sqrt_stub, &sqrt_kernel_cuda);
REGISTER_DISPATCH(trunc_stub, &trunc_kernel_cuda);
REGISTER_DISPATCH(erfinv_stub, &erfinv_kernel_cuda);
REGISTER_DISPATCH(digamma_stub, &digamma_kernel_cuda);
REGISTER_DISPATCH(polygamma_stub, &polygamma_kernel_cuda);
REGISTER_DISPATCH(lgamma_stub, &lgamma_kernel_cuda);
}}
